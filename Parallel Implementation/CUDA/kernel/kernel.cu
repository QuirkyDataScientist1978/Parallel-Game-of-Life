#include "hip/hip_runtime.h"
#include <stdio.h>
#include"hip/hip_runtime.h"
#include""
#include <stdlib.h>
#include<time.h>
#include<iostream>
#include "optimized_kernel.h"
#include "simple_kernel.h"
#include "natural_indexed_kernel.h"
using namespace std;
//int SIZE, ITERATIONS, ANIMATE, BLOCKS, THREADS, SEED, UNOPTIMIZED, PRINT;
void print_board(int board[], int size, int iteration)
{
	if (iteration != -1)
	{
		printf("Iteration %d\n", iteration);
	}
	for (int i = 0;i < size; i++)
	{
		for (int j = 0; j < size; j++)
		{
			if (board[i * size + j] != 0 && board[i * size + j] != 1)
			{
				printf("%d", board[i * size + j]);
			}
			else
			{
				if (board[i * size + j])
				{
					cout << "*";
				}
				else
				{
					cout << "#";
				}
			}
		}
		printf("\n");
	}
	printf("\n\n");
}

int run()
{
	bool animate =true;
	int size =  20;
	int iterations = 200;
	int no_blocks = size;
	int no_threads =  size;
	int unoptimized_run =  1;
	bool print =  true;

	// Initialize random seed
	srand(time(NULL));

	// Allocate space on host
	int *input = (int*)calloc(size * size, sizeof(int));
	int *output = (int*)calloc(size * size, sizeof(int));
	int *devin, *devout, *devtemp;

	// Allocate space on device
	hipMalloc((void**)&devin, size * size * sizeof(int));
	hipMalloc((void**)&devout, size * size * sizeof(int));
	hipMalloc((void**)&devtemp, size * size * sizeof(int));

	// Generate random input
	for (int i = 0;i < size; i++)
	{
		for (int j = 0; j < size; j++)
		{
			input[i*size + j] = rand() % 2;
			//printf("%d", rand() % 2);
		}
	}

	//timing events
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	if (print)
		print_board(input, size, 0);

	// Copy from host to device
	hipMemcpy(devin, input, size * size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(devout, output, size * size * sizeof(int), hipMemcpyHostToDevice);

	int shared_board_size = (no_threads + 2 * size) * sizeof(int);
	// Call the chosen kernel and time the run
	//struct timeval  tv1, tv2;
	//gettimeofday(&tv1, NULL);
	clock_t time = clock();
	if (unoptimized_run)
	{
		printf(" kernel run\n");
		for (int i = 0;i<iterations;i++)
		{
			if (i == 0)
			{
				play_with_row_based_index << <no_blocks, no_threads >> > (devin, devout, size);
			}
			else
			{
				play_with_row_based_index<<<no_blocks, no_threads>>>(devtemp, devout, size);
			}
			hipMemcpy(devtemp, devout, size * size * sizeof(int), hipMemcpyDeviceToDevice);
			hipMemcpy(output, devout, size * size * sizeof(int), hipMemcpyDeviceToHost);
			if (animate == true)
			{
				system("clear");
				print_board(output, size, i);
				_sleep(100);
			}
		}
	}
	

	hipMemcpy(output, devout, size * size * sizeof(int), hipMemcpyDeviceToHost);
	if (print)
		print_board(output, size, iterations);
	printf("Total time in kernel = %f milliseconds\n", (float)(clock() - time) / CLOCKS_PER_SEC);
	hipFree(devin);
	hipFree(devout);
	hipFree(devtemp);

	return 0;
}

int main(int argc, char* argv[])
{
	run();
	return 0;
}
